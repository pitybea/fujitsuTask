#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "main.h"

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {

        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {

        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {

        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {

        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {

        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {

        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {

        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {

        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


int test()
{
	int count;
	hipGetDeviceCount(&count);
	printf("%d\n",count);
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
       
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
       
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.

__global__ void adda(int count,int* a,int *b,int* c)
{
	
	int i= blockDim.x* blockIdx.x +threadIdx.x;
	if(i<count)
		c[i]=a[i]+b[i];
}

void launch(int* a,int* b,int* c,int testsize)
{
	int threadsize=256;
	int blocksize=256;

	adda<<<blocksize,threadsize>>>(testsize,a,b,c);

	hipDeviceSynchronize();
}

int main()
{
	int testsize=100000;
	int* a;
	int* b;

	//a=new int[testsize];
	//b=new int[testsize];

	hipMallocManaged(&a,sizeof(int)*testsize);
	hipMallocManaged(&b,sizeof(int)*testsize);

	for (int i = 0; i < testsize; i++)
	{
		a[i]=i;
		b[i]=testsize-i;
	}
	int* c;

	//c=new int[testsize];
	hipMallocManaged((void**)&c,sizeof(int)*testsize);
	launch(a,b,c,testsize);

	for (int i = 0; i < testsize; i++)
	{
		if(testsize%10==0)
			printf("%d ",c[i]);
			
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);

	hipDeviceReset;
    return 0;
}

__global__ void printValue( int *value) {
	++value[blockIdx.x];
}
 
void hostFunction(int *value){
 
	value[0]=1;
	value[1]=2;
	printValue<<< 2, 1 >>>(value);
	hipDeviceSynchronize();
	hipFree(value);
}
 
int amain() {
	int *value;
	hipMallocManaged(&value, 2 * sizeof(int));

	hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {

       // goto Error;
    }
	hostFunction(value);
	return 0;
}



